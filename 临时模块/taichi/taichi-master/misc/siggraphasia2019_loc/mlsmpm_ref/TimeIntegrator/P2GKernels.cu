#include "hip/hip_runtime.h"
#include "P2GKernels.cuh"
#include <System/CudaDevice/CudaDeviceUtils.cuh>
#include <cstdio>
namespace mn {
__global__ void P2G_MLS(const int numParticle,
                        const int *d_targetPages,
                        const int *d_virtualPageOffsets,
                        const int **smallest_nodes,
                        const T *d_sigma,
                        int *d_block_offsets,
                        int *d_cellids,
                        int *d_indices,
                        int *d_indexTrans,
                        T **d_sorted_positions,
                        T *d_sorted_masses,
                        T **d_sorted_velocities,
                        T *d_B,
                        const T dt,
                        T **d_channels,
                        int **d_adjPage,
                        uint64_t *d_pageOffsets) {
  __shared__ T buffer[4][8][8][8];
  int cellid = (4 * 8 * 8 * 8 + blockDim.x - 1) / blockDim.x;
  for (int i = 0; i < cellid; ++i)
    if (blockDim.x * i + threadIdx.x < 4 * 8 * 8 * 8)
      *((&buffer[0][0][0][0]) + blockDim.x * i + threadIdx.x) = (T)0;
  __syncthreads();
  int pageid = d_targetPages[blockIdx.x] - 1;
  cellid = d_block_offsets[pageid];
  int relParid =
      512 * (blockIdx.x - d_virtualPageOffsets[pageid]) + threadIdx.x;
  int parid = cellid + relParid;
  int laneid = threadIdx.x & 0x1f;
  bool bBoundary;
  if (relParid < d_block_offsets[pageid + 1] - d_block_offsets[pageid]) {
    cellid = d_cellids[parid] - 1;
    bBoundary = laneid == 0 || cellid + 1 != d_cellids[parid - 1];
  } else
    bBoundary = true;
  uint32_t mark = __ballot(bBoundary);  // a bit-mask
  mark = __brev(mark);
  unsigned int interval = min(__clz(mark << (laneid + 1)), 31 - laneid);
  mark = interval;
  for (int iter = 1; iter & 0x1f; iter <<= 1) {
    int tmp = __shfl_down(mark, iter);
    mark = tmp > mark ? tmp : mark; /*if (tmp > mark) mark = tmp;*/
  }
  mark = __shfl(mark, 0);
  __syncthreads();
  int smallest_node[3];
  if (relParid < d_block_offsets[pageid + 1] - d_block_offsets[pageid]) {
    T wOneD[3][3];
    smallest_node[0] = smallest_nodes[0][cellid];
    smallest_node[1] = smallest_nodes[1][cellid];
    smallest_node[2] = smallest_nodes[2][cellid];
    int parid_trans = d_indexTrans[parid];
    T sig[9];
    sig[0] = d_sigma[parid_trans + (0) * numParticle];
    sig[1] = d_sigma[parid_trans + (1) * numParticle];
    sig[2] = d_sigma[parid_trans + (2) * numParticle];
    sig[3] = d_sigma[parid_trans + (3) * numParticle];
    sig[4] = d_sigma[parid_trans + (4) * numParticle];
    sig[5] = d_sigma[parid_trans + (5) * numParticle];
    sig[6] = d_sigma[parid_trans + (6) * numParticle];
    sig[7] = d_sigma[parid_trans + (7) * numParticle];
    sig[8] = d_sigma[parid_trans + (8) * numParticle];
    T B[9];
    B[0] = d_B[parid_trans + 0 * numParticle];
    B[1] = d_B[parid_trans + 1 * numParticle];
    B[2] = d_B[parid_trans + 2 * numParticle];
    B[3] = d_B[parid_trans + 3 * numParticle];
    B[4] = d_B[parid_trans + 4 * numParticle];
    B[5] = d_B[parid_trans + 5 * numParticle];
    B[6] = d_B[parid_trans + 6 * numParticle];
    B[7] = d_B[parid_trans + 7 * numParticle];
    B[8] = d_B[parid_trans + 8 * numParticle];
    T mass = d_sorted_masses[d_indices[parid]];
    for (int i = 0; i < 9; ++i)
      B[i] = (B[i] * mass - sig[i] * dt) * D_inverse;
    T xp[3];
    xp[0] = d_sorted_positions[0][parid] - smallest_node[0] * dx;
    xp[1] = d_sorted_positions[1][parid] - smallest_node[1] * dx;
    xp[2] = d_sorted_positions[2][parid] - smallest_node[2] * dx;
    for (int v = 0; v < 3; ++v) {
      T d0 = xp[v] * one_over_dx;
      T z = ((T)1.5 - d0);
      wOneD[v][0] = (T)0.5 * z * z;
      d0 = d0 - 1.0f;
      wOneD[v][1] = (T)0.75 - d0 * d0;
      z = (T)1.5 - (1.0f - d0);
      wOneD[v][2] = (T)0.5 * z * z;
    }
    T vel[3];
    vel[0] = d_sorted_velocities[0][parid_trans];
    vel[1] = d_sorted_velocities[1][parid_trans];
    vel[2] = d_sorted_velocities[2][parid_trans];
    smallest_node[0] = smallest_node[0] & 0x3;
    smallest_node[1] = smallest_node[1] & 0x3;
    smallest_node[2] = smallest_node[2] & 0x3;
    T val[4];
    for (int i = 0; i < 3; ++i) {
      for (int j = 0; j < 3; ++j) {
        for (int k = 0; k < 3; ++k) {
          T weight = wOneD[0][i] * wOneD[1][j] * wOneD[2][k];
#ifdef DEBUG_INFO
          if (weight < 0 || weight > 1)
            printf("weight is negative!!! %f\n", weight);
#endif
          val[0] = mass * weight;
          T xi_minus_xp[3];
          xi_minus_xp[0] = i * dx - xp[0];
          xi_minus_xp[1] = j * dx - xp[1];
          xi_minus_xp[2] = k * dx - xp[2];
          val[1] = val[0] * vel[0];
          val[2] = val[0] * vel[1];
          val[3] = val[0] * vel[2];
          val[1] += (B[0] * xi_minus_xp[0] + B[3] * xi_minus_xp[1] +
                     B[6] * xi_minus_xp[2]) *
                    weight;
          val[2] += (B[1] * xi_minus_xp[0] + B[4] * xi_minus_xp[1] +
                     B[7] * xi_minus_xp[2]) *
                    weight;
          val[3] += (B[2] * xi_minus_xp[0] + B[5] * xi_minus_xp[1] +
                     B[8] * xi_minus_xp[2]) *
                    weight;
          for (int iter = 1; iter <= mark; iter <<= 1) {
            T tmp[4];
            for (int i = 0; i < 4; ++i)
              tmp[i] = __shfl_down(val[i], iter);
            if (interval >= iter)
              for (int i = 0; i < 4; ++i)
                val[i] += tmp[i];
          }
          if (bBoundary)
            for (int ii = 0; ii < 4; ++ii)
              atomicAdd(&(buffer[ii][smallest_node[0] + i][smallest_node[1] + j]
                                [smallest_node[2] + k]),
                        val[ii]);
        }
      }
    }
  }
  __syncthreads();
  int block = threadIdx.x & 0x3f;
  int ci = block >> 4;
  int cj = (block & 0xc) >> 2;
  int ck = block & 3;
  block = threadIdx.x >> 6;
  int bi = block >> 2;
  int bj = (block & 2) >> 1;
  int bk = block & 1;
  int page_idx = block ? d_adjPage[block - 1][pageid] : pageid;
  for (int ii = 0; ii < 4; ++ii)
    if (buffer[ii][bi * 4 + ci][bj * 4 + cj][bk * 4 + ck] != 0)
      atomicAdd((T *)((uint64_t)d_channels[ii] + page_idx * 4096) +
                    (ci * 16 + cj * 4 + ck),
                buffer[ii][bi * 4 + ci][bj * 4 + cj][bk * 4 + ck]);
}
}  // namespace mn
